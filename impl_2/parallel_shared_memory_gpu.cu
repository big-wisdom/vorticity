#include "hip/hip_runtime.h"
#include <cstring> 
#include <fstream>
#include <iostream>
#include <vector>
#include "vorticity.hpp"


#define WIDTH 1300
#define HEIGHT 600
#define BLOCK_WIDTH 20
#define BLOCK_HEIGHT 30
#define TILE_WIDTH 65
#define TILE_HEIGHT 20
#define HALO 2
#define CHANNELS 2

//This doesn't work just the beginnings of an idea

__global__
void convertTile(int height, int width, unsigned char *output, float *input) {

  __shared__ float vortTile[TILE_WIDTH + HALO][TILE_HEIGHT + HALO][CHANNELS];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  // Copy over the vector information to the tile
  // Copy over the vector information to the tile
  
  if (x % TILE_WIDTH == 0 && x != 0) {
    vortTile[threadIdx.x][threadIdx.y][0] = input[CHANNELS * (y * WIDTH + (x - 1))];
    vortTile[threadIdx.x][threadIdx.y][1] = input[CHANNELS * (y * WIDTH + (x - 1)) + 1];
  }
  
  if (x % (TILE_WIDTH - 1) == 0 && x != width - 1) {
    vortTile[threadIdx.x + 2][threadIdx.y][0] = input[CHANNELS * (y * WIDTH + (x + 1))];
    vortTile[threadIdx.x + 2][threadIdx.y][1] = input[CHANNELS * (y * WIDTH + (x + 1)) + 1];
  }
 
  if (y % TILE_HEIGHT == 0 && y != 0) {
    vortTile[threadIdx.x][threadIdx.y][0] = input[CHANNELS * ((y - 1) * WIDTH + x)];
    vortTile[threadIdx.x][threadIdx.y][1] = input[CHANNELS * ((y - 1) * WIDTH + x) + 1];
  }
  
  if (y % (TILE_HEIGHT - 1) == 0 && y != height - 1) {
    vortTile[threadIdx.x][threadIdx.y + 2][0] = input[CHANNELS * ((y + 1) * WIDTH + x)];
    vortTile[threadIdx.x][threadIdx.y + 2][1] = input[CHANNELS * ((y + 1) * WIDTH + x) + 1];
  }
  vortTile[threadIdx.x + 1][threadIdx.y + 1][0] = input[CHANNELS * (y * WIDTH + x)];
  vortTile[threadIdx.x + 1][threadIdx.y + 1][1] = input[CHANNELS * (y * WIDTH + x) + 1];
  __syncthreads();

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : threadIdx.x;
  int end_x = (x == width - 1) ? threadIdx.x + 1: threadIdx.x + 2;

  int start_y = (y == 0) ? 0 : threadIdx.y;
  int end_y = (y == height - 1) ? threadIdx.y + 1: threadIdx.y + 2;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {vortTile[end_x][start_y][0], vortTile[end_x][start_y][1]};
  double fdv[2] = {vortTile[start_x][end_y][0], vortTile[start_x][end_y][1]};
  double vec0[2] = {vortTile[threadIdx.x + 1][threadIdx.y + 1][0], vortTile[threadIdx.x + 1][threadIdx.y + 1][1]};
  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function
  unsigned char vortChar;
    if (vort < -0.2f) {
      vortChar = 0;
    } else if (vort > 0.2f) {
      vortChar = 127;
    } else {
      vortChar = 255;
    }
    output[y * width + x] = vortChar;
    __syncthreads();

}

__global__
void convert(int height, int width, unsigned char *output, float *input) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : x - 1;
  int end_x = (x == width - 1) ? x : x + 1;

  int start_y = (y == 0) ? 0 : y - 1;
  int end_y = (y == height - 1) ? y : y + 1;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {input[duidx], input[duidx + 1]};
  double fdv[2] = {input[dvidx], input[dvidx + 1]};
  double vec0[2] = {input[idx * 2], input[idx * 2 + 1]};

  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function 

  unsigned char vortChar;
  if (vort < -0.2f) {
    vortChar = 0;
  } else if (vort > 0.2f) {
    vortChar = 127;
  } else {
    vortChar = 255;
  }
  output[y * width + x] = vortChar;
}

void parallel_shared_memory_gpu(int height, int width, float * input, unsigned char * output, int length) {
    //Prepare cuda stuff
    float *inputDevice;
    unsigned char * outputDevice;
    hipMalloc((void **) &inputDevice, length);
    hipMalloc((void **) &outputDevice, length);

    hipMemcpy(inputDevice, input, length, hipMemcpyHostToDevice);
    hipMemcpy(outputDevice, output, length, hipMemcpyHostToDevice);

    const dim3 block_size (TILE_WIDTH, TILE_HEIGHT);
    const dim3 grid_size (BLOCK_WIDTH, BLOCK_HEIGHT);

    convertTile<<<block_size, grid_size>>>(height, width, outputDevice, inputDevice);
    printf("Error: %d", hipDeviceSynchronize());

    //Return image to device and free memory
    hipMemcpy(output, outputDevice, length, hipMemcpyDeviceToHost);
    hipFree(inputDevice);
    hipFree(outputDevice);
}


int main() {
  std::ifstream vectorField("cyl2d_1300x600_float32[2].raw", std::ios::binary);
  if (vectorField.is_open()) {
    // Get the length of the image should be 3145728
    std::cout << "opened" << std::endl;
    vectorField.seekg(0, std::ios_base::end);
    auto length = vectorField.tellg();
    vectorField.seekg(0, std::ios::beg);

    // Initialize arrays
    float *input = new float[length];
    unsigned char *output = new unsigned char[length / CHANNELS];

    // Get rgb values from image into input array
    vectorField.read((char *)input, length);
    vectorField.close();

    // serial_vorticity(HEIGHT, WIDTH, input, output);
    //parallel_shared_memory_cpu(HEIGHT, WIDTH, input, output);
    parallel_shared_memory_gpu(HEIGHT, WIDTH, input, output, length);
    // Writing output to file
    std::fstream outField("outfield.raw", std::ios::out | std::ios::binary);
    outField.write(reinterpret_cast<char *>(output), length / CHANNELS);
    outField.close();
  } else {
    std::cout << "Didn't open" << std::endl;
  }
  return 0;
}
