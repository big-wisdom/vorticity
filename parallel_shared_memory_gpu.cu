#include "hip/hip_runtime.h"
#include <cstring> 
#include <fstream>
#include <iostream>
#include <vector>
#include "vorticity.hpp"


#define WIDTH 1300
#define HEIGHT 600
#define GRID_WIDTH 20
#define GRID_HEIGHT 30
#define BLOCK_WIDTH 65
#define BLOCK_HEIGHT 20
#define HALO 2
#define CHANNELS 2

//This doesn't work just the beginnings of an idea

__global__
void convertTile(int height, int width, unsigned char *output, float *input) {

  __shared__ float vortTile[BLOCK_WIDTH + HALO][BLOCK_HEIGHT + HALO][CHANNELS];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  // Copy over the vector information to the tile
  
  if (threadIdx.x == 0 && x != 0) {
    vortTile[threadIdx.x][threadIdx.y + 1][0] = input[CHANNELS * (y * WIDTH + (x - 1))];
    vortTile[threadIdx.x][threadIdx.y + 1][1] = input[CHANNELS * (y * WIDTH + (x - 1)) + 1];
  }
  
  if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) {
    vortTile[threadIdx.x + 2][threadIdx.y + 1][0] = input[CHANNELS * (y * WIDTH + (x + 1))];
    vortTile[threadIdx.x + 2][threadIdx.y + 1][1] = input[CHANNELS * (y * WIDTH + (x + 1)) + 1];
  }
 
  if (threadIdx.y == 0 && y != 0) {
    vortTile[threadIdx.x + 1][threadIdx.y][0] = input[CHANNELS * ((y - 1) * WIDTH + x)];
    vortTile[threadIdx.x + 1][threadIdx.y][1] = input[CHANNELS * ((y - 1) * WIDTH + x) + 1];
  }
  
  if (threadIdx.y == (BLOCK_HEIGHT - 1) && y != height - 1) {
    vortTile[threadIdx.x + 1][threadIdx.y + 2][0] = input[CHANNELS * ((y + 1) * WIDTH + x)];
    vortTile[threadIdx.x + 1][threadIdx.y + 2][1] = input[CHANNELS * ((y + 1) * WIDTH + x) + 1];
  }
  vortTile[threadIdx.x + 1][threadIdx.y + 1][0] = input[CHANNELS * (y * WIDTH + x)];
  vortTile[threadIdx.x + 1][threadIdx.y + 1][1] = input[CHANNELS * (y * WIDTH + x) + 1];
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0 && x == 0 && y == 0)
  {
    for(int i = 1; i < BLOCK_WIDTH + 1; i++) {
      for(int j = 1; j < BLOCK_HEIGHT + 1; j++) {
        int newX = (i - 1) + blockIdx.x * blockDim.x;
        int newY = (j - 1) + blockIdx.y * blockDim.y;
        printf("x: %d y: %d Tile x: %f Tile y: %f input x: %f input y: %f\n",newX, newY,vortTile[i][j][0], vortTile[i][j][1], input[CHANNELS * (newY * width + (newX))], input[CHANNELS * (newY * width + (newX)) + 1]);
      }
    }
  }

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  //uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : threadIdx.x;
  int end_x = (x == width - 1) ? threadIdx.x + 1: threadIdx.x + 2;

  int start_y = (y == 0) ? 0 : threadIdx.y;
  int end_y = (y == height - 1) ? threadIdx.y + 1: threadIdx.y + 2;

  // duidx = (start_y * width + end_x) * 2;
  //uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {vortTile[end_x][start_y][0], vortTile[end_x][start_y][1]};
  double fdv[2] = {vortTile[start_x][end_y][0], vortTile[start_x][end_y][1]};
  double vec0[2] = {vortTile[threadIdx.x + 1][threadIdx.y + 1][0], vortTile[threadIdx.x + 1][threadIdx.y + 1][1]};
  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function
  unsigned char vortChar;
    if (vort < -0.2f) {
      vortChar = 0;
    } else if (vort > 0.2f) {
      vortChar = 127;
    } else {
      vortChar = 255;
    }
    output[y * width + x] = vortChar;
    __syncthreads();

}

__global__
void convert(int height, int width, unsigned char *output, float *input) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : x - 1;
  int end_x = (x == width - 1) ? x : x + 1;

  int start_y = (y == 0) ? 0 : y - 1;
  int end_y = (y == height - 1) ? y : y + 1;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {input[duidx], input[duidx + 1]};
  double fdv[2] = {input[dvidx], input[dvidx + 1]};
  double vec0[2] = {input[idx * 2], input[idx * 2 + 1]};

  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function 

  unsigned char vortChar;
  if (vort < -0.2f) {
    vortChar = 0;
  } else if (vort > 0.2f) {
    vortChar = 127;
  } else {
    vortChar = 255;
  }
  output[y * width + x] = vortChar;
}

void parallel_shared_memory_gpu(int height, int width, float * input, unsigned char * output, int length) {
    //Prepare cuda stuff
    float *inputDevice;
    unsigned char * outputDevice;
    hipMalloc((void **) &inputDevice, length);
    hipMalloc((void **) &outputDevice, length / 8);

    hipMemcpy(inputDevice, input, length, hipMemcpyHostToDevice);
    hipMemcpy(outputDevice, output, length / 8, hipMemcpyHostToDevice);

    const dim3 block_size (BLOCK_WIDTH, BLOCK_HEIGHT);
    const dim3 grid_size (GRID_WIDTH, GRID_HEIGHT);

    convertTile<<<block_size, grid_size>>>(height, width, outputDevice, inputDevice);
    printf("Error: %d", hipDeviceSynchronize());

    //Return image to device and free memory
    hipMemcpy(output, outputDevice, length / 8, hipMemcpyDeviceToHost);
    hipFree(inputDevice);
    hipFree(outputDevice);
}


int main() {
  std::ifstream vectorField("cyl2d_1300x600_float32[2].raw", std::ios::binary);
  if (vectorField.is_open()) {
    // Get the length of the image should be 3145728
    std::cout << "opened" << std::endl;
    vectorField.seekg(0, std::ios_base::end);
    auto length = vectorField.tellg();
    printf("%d", length);
    vectorField.seekg(0, std::ios::beg);

    auto fl_size = sizeof(float);

    // Initialize arrays
    float *input = new float[length / fl_size];
    unsigned char *output = new unsigned char[length / fl_size / CHANNELS];

    // Get rgb values from image into input array
    vectorField.read((char *)input, length);
    vectorField.close();

    // serial_vorticity(HEIGHT, WIDTH, input, output);
    //parallel_shared_memory_cpu(HEIGHT, WIDTH, input, output);
    parallel_shared_memory_gpu(HEIGHT, WIDTH, input, output, length);
    // Writing output to file
    std::fstream outField("outfield.raw", std::ios::out | std::ios::binary);
    outField.write(reinterpret_cast<char *>(output), length / CHANNELS);
    outField.close();
  } else {
    std::cout << "Didn't open" << std::endl;
  }
  return 0;
}
