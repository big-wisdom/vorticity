#include "hip/hip_runtime.h"
#include <cstring> 
#include <fstream>
#include <iostream>
#include <vector>
#include "vorticity.hpp"


#define WIDTH 1300
#define HEIGHT 600
#define GRID_WIDTH 65
#define GRID_HEIGHT 20
#define BLOCK_WIDTH 20
#define BLOCK_HEIGHT 30
#define HALO 2
#define CHANNELS 2

//This doesn't work just the beginnings of an idea

__global__
void convertTile(int height, int width, unsigned char *output, float *input) {

  __shared__ float vortTile[BLOCK_WIDTH + HALO][BLOCK_HEIGHT + HALO][CHANNELS];

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);

  // Copy over the vector information to the tile
  
  if (threadIdx.x == 0 && x != 0) { //get Left Halo
    vortTile[threadIdx.x][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x - 1))];
    vortTile[threadIdx.x][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x - 1))) + 1];
  }
  
  if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Right Halo
    vortTile[threadIdx.x + 2][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x + 1))];
    vortTile[threadIdx.x + 2][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x + 1))) + 1];
  }
 
  if (threadIdx.y == 0 && y != 0) { //get Upper Halo
    if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Upper Right Corner 
      vortTile[threadIdx.x + 2][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1))];
      vortTile[threadIdx.x + 2][threadIdx.y][1] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1)) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + x)];
    vortTile[threadIdx.x + 1][threadIdx.y][1] = input[(CHANNELS * (((y - 1) * WIDTH) + x)) + 1];
  }
  
  if (threadIdx.y == (BLOCK_HEIGHT - 1) && y != height - 1) { // Get Lower Halo
    if (threadIdx.x == 0 && x != 0) { //get Lower Left Corner
      vortTile[threadIdx.x][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + (x - 1))];  
      vortTile[threadIdx.x][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + (x - 1))) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + x)];  
    vortTile[threadIdx.x + 1][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + x)) + 1];
  }
  vortTile[threadIdx.x + 1][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + x)];
  vortTile[threadIdx.x + 1][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + x)) + 1];
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0)
  {
    for(int i = 0; i < 32; i ++){
      for (int j = 0; j < 22; j++) {
        int zeroX = 0 + (blockIdx.x * blockDim.x);
        int zeroY = 0 + (blockIdx.y * blockDim.y);
        int newX = zeroX + j - 1;
        int newY = zeroY + i - 1;
        if( (i != 0 || j != 0) && (j != 21 || i != 0) && (j != 0 || i != 31) && (j != 21 || i != 31)) {
          if (newX > -1 && newX < width && newY > -1 && newY < height ) {
            if (vortTile[j][i][0] != input[2 * ((newY * width) + newX)] || vortTile[j][i][1] != input[2 * ((newY * width) + newX) + 1]) {
              printf("x: %d y: %d Tile x: %f Tile y: %f input x: %f input y: %f\n",newX, newY,vortTile[j][i][0], vortTile[j][i][1], input[2 * ((newY * width) + newX)], input[2 * ((newY * width) + newX) + 1]);
            }
          }
        }
      }
    }
  }

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 1 : threadIdx.x;
  int end_x = (x == width - 1) ? threadIdx.x + 1: threadIdx.x + 2;

  int start_y = (y == 0) ? 1 : threadIdx.y;
  int end_y = (y == height - 1) ? threadIdx.y + 1: threadIdx.y + 2;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {vortTile[end_x][start_y][0], vortTile[end_x][start_y][1]};
  double fdv[2] = {vortTile[start_x][end_y][0], vortTile[start_x][end_y][1]};
  double vec0[2] = {vortTile[threadIdx.x + 1][threadIdx.y + 1][0], vortTile[threadIdx.x + 1][threadIdx.y + 1][1]};
  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function

  // add old vorticity function and run on input compare outputs and see if vorticity is messing something up 
  //The vorticity funciton
  dx = 0.01;
  dy = 0.01;

  idx = y * width + x;

  start_x = (x == 0) ? 0 : x - 1;
  end_x = (x == width - 1) ? x : x + 1;

  start_y = (y == 0) ? 0 : y - 1;
  end_y = (y == height - 1) ? y : y + 1;

  duidx = (start_y * width + end_x) * 2;
  dvidx = (end_y * width + start_x) * 2;

  double fdu2[2] = {input[duidx], input[duidx + 1]};
  double fdv2[2] = {input[dvidx], input[dvidx + 1]};
  double vec02[2] = {input[idx * 2], input[idx * 2 + 1]};

  float duy2 = (fdu2[1] - vec02[1]) / (dx * (end_x - start_x));
  float dvx2 = (fdv2[0] - vec02[0]) / (dy * (end_y - start_y));

  float vort2 = duy2 - dvx2;
  //End of vorticity function 
  if (dvx != dvx2 && blockIdx.x == 1 and blockIdx.y == 0) {
    printf("problem tile:%f original: %f at x: %d y: %d\n", dvx, dvx2, x, y);
  }
  if (fdv[0] != fdv2[0] && blockIdx.x == 1 and blockIdx.y == 0) {
    printf("fdv problem tile:%f original: %f at x: %d y: %d\n", fdv[0], fdv2[0], x, y);
  }

  unsigned char vortChar;
    if (vort < -0.2f) {
      vortChar = 0;
    } else if (vort > 0.2f) {
      vortChar = 127;
    } else {
      vortChar = 255;
    }
    output[y * width + x] = vortChar;
    __syncthreads();

}

__global__
void convert(int height, int width, unsigned char *output, float *input) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : x - 1;
  int end_x = (x == width - 1) ? x : x + 1;

  int start_y = (y == 0) ? 0 : y - 1;
  int end_y = (y == height - 1) ? y : y + 1;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {input[duidx], input[duidx + 1]};
  double fdv[2] = {input[dvidx], input[dvidx + 1]};
  double vec0[2] = {input[idx * 2], input[idx * 2 + 1]};

  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function 

  unsigned char vortChar;
  if (vort < -0.2f) {
    vortChar = 0;
  } else if (vort > 0.2f) {
    vortChar = 127;
  } else {
    vortChar = 255;
  }
  output[y * width + x] = vortChar;
}

void parallel_shared_memory_gpu(int height, int width, float * input, unsigned char * output, int length) {
    //Prepare cuda stuff
    float *inputDevice;
    unsigned char * outputDevice;
    hipMalloc((void **) &inputDevice, length);
    hipMalloc((void **) &outputDevice, length / 8);

    hipMemcpy(inputDevice, input, length, hipMemcpyHostToDevice);
    hipMemcpy(outputDevice, output, length / 8, hipMemcpyHostToDevice);

    const dim3 block_size (BLOCK_WIDTH, BLOCK_HEIGHT);
    const dim3 grid_size (GRID_WIDTH, GRID_HEIGHT);

    convertTile<<<grid_size, block_size>>>(height, width, outputDevice, inputDevice);
    printf("Error: %d", hipDeviceSynchronize());

    //Return image to device and free memory
    hipMemcpy(output, outputDevice, length / 8, hipMemcpyDeviceToHost);
    hipFree(inputDevice);
    hipFree(outputDevice);
}


int main() {
  std::ifstream vectorField("cyl2d_1300x600_float32[2].raw", std::ios::binary);
  if (vectorField.is_open()) {
    // Get the length of the image should be 3145728
    std::cout << "opened" << std::endl;
    vectorField.seekg(0, std::ios_base::end);
    auto length = vectorField.tellg();
    vectorField.seekg(0, std::ios::beg);

    auto fl_size = sizeof(float);

    // Initialize arrays
    float *input = new float[length / fl_size];
    unsigned char *output = new unsigned char[length / fl_size / CHANNELS];

    // Get rgb values from image into input array
    vectorField.read((char *)input, length);
    vectorField.close();

    // serial_vorticity(HEIGHT, WIDTH, input, output);
    //parallel_shared_memory_cpu(HEIGHT, WIDTH, input, output);
    parallel_shared_memory_gpu(HEIGHT, WIDTH, input, output, length);
    // Writing output to file
    std::fstream outField("outfield.raw", std::ios::out | std::ios::binary);
    outField.write(reinterpret_cast<char *>(output), length / CHANNELS);
    outField.close();
  } else {
    std::cout << "Didn't open" << std::endl;
  }
  return 0;
}
