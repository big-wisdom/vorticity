/*
  This code is meant to be run on a cuda device. 
  You must first use the command module load cuda 
  to ensure that you can compile correctly with 
  nvcc parallel_shared_memory_gpu.cu -o object_name.
  Once the code is compiled you can run then run the
  code with ./object_name assuming you have prepared 
  the gpu correctly. 
*/

#include <hip/hip_runtime.h>
#include <cstring> 
#include <iostream>
#include <string>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}



#define WIDTH 1300
#define HEIGHT 600
#define GRID_WIDTH 65
#define GRID_HEIGHT 20
#define BLOCK_WIDTH 20
#define BLOCK_HEIGHT 30
// #define WIDTH 260
// #define HEIGHT 120
// #define GRID_WIDTH 13
// #define GRID_HEIGHT 4
// #define BLOCK_WIDTH 20
// #define BLOCK_HEIGHT 30
#define HALO 2
#define CHANNELS 2

__global__
void convertTile(int height, int width, unsigned char *output, float *input, int my_rank, int core_count) {

  __shared__ float vortTile[BLOCK_WIDTH + HALO][BLOCK_HEIGHT + HALO][CHANNELS];

  // these are coordinates in the already tiled block that the node recieved
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  // adjust those coordinates for the data block this node is responsible for
  if (my_rank != 0) y+=1;

  // Copy over the vector information to the tile
  if (threadIdx.x == 0 && x != 0) { //get Left Halo
    vortTile[threadIdx.x][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x - 1))];
    vortTile[threadIdx.x][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x - 1))) + 1];
  }
  
  if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Right Halo
    vortTile[threadIdx.x + 2][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x + 1))];
    vortTile[threadIdx.x + 2][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x + 1))) + 1];
  }
 
  if (threadIdx.y == 0 && y != 0) { //get Upper Halo
    if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Upper Right Corner 
      vortTile[threadIdx.x + 2][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1))];
      vortTile[threadIdx.x + 2][threadIdx.y][1] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1)) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + x)];
    vortTile[threadIdx.x + 1][threadIdx.y][1] = input[(CHANNELS * (((y - 1) * WIDTH) + x)) + 1];
  }
  
  if (threadIdx.y == (BLOCK_HEIGHT - 1) && y != height - 1) { // Get Lower Halo
    if (threadIdx.x == 0 && x != 0) { //get Lower Left Corner
      vortTile[threadIdx.x][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + (x - 1))];  
      vortTile[threadIdx.x][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + (x - 1))) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + x)];  
    vortTile[threadIdx.x + 1][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + x)) + 1];
  }
  vortTile[threadIdx.x + 1][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + x)];
  vortTile[threadIdx.x + 1][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + x)) + 1];
  __syncthreads();

  //I am not sure if cuda can call a function that is in another file so I just put this here. 
  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  int start_x = (x == 0) ? 1 : threadIdx.x;
  int end_x = (x == width - 1) ? threadIdx.x + 1: threadIdx.x + 2;

  int start_y = (y == 0) ? 1 : threadIdx.y;
  int end_y = (y == height - 1) ? threadIdx.y + 1: threadIdx.y + 2;

  double fdu[2] = {vortTile[end_x][start_y][0], vortTile[end_x][start_y][1]};
  double fdv[2] = {vortTile[start_x][end_y][0], vortTile[start_x][end_y][1]};
  double vec0[2] = {vortTile[threadIdx.x + 1][threadIdx.y + 1][0], vortTile[threadIdx.x + 1][threadIdx.y + 1][1]};
  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function

  unsigned char vortChar;
  if (vort < -0.2f) {
    vortChar = 0;
  } else if (vort > 0.2f) {
    vortChar = 127;
  } else {
    vortChar = 255;
  }
  output[y * width + x] = vortChar;
  __syncthreads();

}


extern "C" void parallel_shared_memory_gpu(int height, int width, float* input, unsigned char* output, int length, int my_rank, int core_count) {
    //Prepare cuda stuff
    float *inputDevice;
    unsigned char * outputDevice;
    hipMalloc((void **) &inputDevice, length);
    hipMalloc((void **) &outputDevice, length / 8);

    hipMemcpy(inputDevice, input, length, hipMemcpyHostToDevice);
    hipMemcpy(outputDevice, output, length / 8, hipMemcpyHostToDevice);

    // calculate grid and block size
    int data_height = height - 2; // data size is the height minus the top and bottom
    if (my_rank == 0 || my_rank == core_count - 1) data_height += 1; // unless this is the fist or last rank
    int grid_height = data_height / BLOCK_HEIGHT;
    if (grid_height * BLOCK_HEIGHT < data_height) {
      grid_height += 1;
    }
    printf("Height: %d, data_height: %d grid_height: %d\n", height, data_height, grid_height);
    const dim3 block_size (BLOCK_WIDTH, BLOCK_HEIGHT);
    const dim3 grid_size (GRID_WIDTH, grid_height);

    convertTile<<<grid_size, block_size>>>(height, width, outputDevice, inputDevice, my_rank, core_count);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // printf("Error: %d", cudaDeviceSynchronize());

    //Return image to device and free memory
    hipMemcpy(output, outputDevice, length / 8, hipMemcpyDeviceToHost);
    hipFree(inputDevice);
    hipFree(outputDevice);
}

// int main()
// {
//     int height = 301;
//     int width = 1300;
//     int channels = 2;
// 
//     // read in input
//     float* input = (float*)malloc(height*width*channels*sizeof(float));
//     unsigned char* output = (unsigned char*)malloc(height*width*sizeof(unsigned char));
//     FILE* pf = fopen("cyl2d_1300x600_float32[2].raw", "rb");
//     fread(input, sizeof(float), height*width*channels, pf);
//     fclose(pf);
// 
//     parallel_shared_memory_gpu(height, width, input, output, height*width*channels*sizeof(float), 0, 2);
// }
