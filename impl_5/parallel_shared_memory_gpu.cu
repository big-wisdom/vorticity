/*
  This code is meant to be run on a cuda device. 
  You must first use the command module load cuda 
  to ensure that you can compile correctly with 
  nvcc parallel_shared_memory_gpu.cu -o object_name.
  Once the code is compiled you can run then run the
  code with ./object_name assuming you have prepared 
  the gpu correctly. 
*/


#include <hip/hip_runtime.h>
#include <cstring> 
#include <iostream>
#include <string>


#define WIDTH 1300
#define HEIGHT 600
#define GRID_WIDTH 65
#define GRID_HEIGHT 20
#define BLOCK_WIDTH 20
#define BLOCK_HEIGHT 30
#define HALO 2
#define CHANNELS 2

__global__
void convertTile(int height, int width, unsigned char *output, float *input) {

  __shared__ float vortTile[BLOCK_WIDTH + HALO][BLOCK_HEIGHT + HALO][CHANNELS];

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);

  // Copy over the vector information to the tile
  
  if (threadIdx.x == 0 && x != 0) { //get Left Halo
    vortTile[threadIdx.x][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x - 1))];
    vortTile[threadIdx.x][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x - 1))) + 1];
  }
  
  if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Right Halo
    vortTile[threadIdx.x + 2][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + (x + 1))];
    vortTile[threadIdx.x + 2][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + (x + 1))) + 1];
  }
 
  if (threadIdx.y == 0 && y != 0) { //get Upper Halo
    if (threadIdx.x == (BLOCK_WIDTH - 1) && x != width - 1) { //get Upper Right Corner 
      vortTile[threadIdx.x + 2][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1))];
      vortTile[threadIdx.x + 2][threadIdx.y][1] = input[CHANNELS * (((y - 1) * WIDTH) + (x + 1)) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y][0] = input[CHANNELS * (((y - 1) * WIDTH) + x)];
    vortTile[threadIdx.x + 1][threadIdx.y][1] = input[(CHANNELS * (((y - 1) * WIDTH) + x)) + 1];
  }
  
  if (threadIdx.y == (BLOCK_HEIGHT - 1) && y != height - 1) { // Get Lower Halo
    if (threadIdx.x == 0 && x != 0) { //get Lower Left Corner
      vortTile[threadIdx.x][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + (x - 1))];  
      vortTile[threadIdx.x][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + (x - 1))) + 1];
    }
    vortTile[threadIdx.x + 1][threadIdx.y + 2][0] = input[CHANNELS * (((y + 1) * WIDTH) + x)];  
    vortTile[threadIdx.x + 1][threadIdx.y + 2][1] = input[(CHANNELS * (((y + 1) * WIDTH) + x)) + 1];
  }
  vortTile[threadIdx.x + 1][threadIdx.y + 1][0] = input[CHANNELS * ((y * WIDTH) + x)];
  vortTile[threadIdx.x + 1][threadIdx.y + 1][1] = input[(CHANNELS * ((y * WIDTH) + x)) + 1];
  __syncthreads();

  //I am not sure if cuda can call a function that is in another file so I just put this here. 
  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  int start_x = (x == 0) ? 1 : threadIdx.x;
  int end_x = (x == width - 1) ? threadIdx.x + 1: threadIdx.x + 2;

  int start_y = (y == 0) ? 1 : threadIdx.y;
  int end_y = (y == height - 1) ? threadIdx.y + 1: threadIdx.y + 2;

  double fdu[2] = {vortTile[end_x][start_y][0], vortTile[end_x][start_y][1]};
  double fdv[2] = {vortTile[start_x][end_y][0], vortTile[start_x][end_y][1]};
  double vec0[2] = {vortTile[threadIdx.x + 1][threadIdx.y + 1][0], vortTile[threadIdx.x + 1][threadIdx.y + 1][1]};
  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function

  unsigned char vortChar;
    if (vort < -0.2f) {
      vortChar = 0;
    } else if (vort > 0.2f) {
      vortChar = 127;
    } else {
      vortChar = 255;
    }
    output[y * width + x] = vortChar;
    __syncthreads();

}

//An old global convert function
__global__
void convert(int height, int width, unsigned char *output, float *input) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  //The vorticity funciton
  float dx = 0.01;
  float dy = 0.01;

  uint32_t idx = y * width + x;

  int start_x = (x == 0) ? 0 : x - 1;
  int end_x = (x == width - 1) ? x : x + 1;

  int start_y = (y == 0) ? 0 : y - 1;
  int end_y = (y == height - 1) ? y : y + 1;

  uint32_t duidx = (start_y * width + end_x) * 2;
  uint32_t dvidx = (end_y * width + start_x) * 2;

  double fdu[2] = {input[duidx], input[duidx + 1]};
  double fdv[2] = {input[dvidx], input[dvidx + 1]};
  double vec0[2] = {input[idx * 2], input[idx * 2 + 1]};

  float duy = (fdu[1] - vec0[1]) / (dx * (end_x - start_x));
  float dvx = (fdv[0] - vec0[0]) / (dy * (end_y - start_y));

  float vort = duy - dvx;
  //End of vorticity function 

  unsigned char vortChar;
  if (vort < -0.2f) {
    vortChar = 0;
  } else if (vort > 0.2f) {
    vortChar = 127;
  } else {
    vortChar = 255;
  }
  output[y * width + x] = vortChar;
}

extern "C" void parallel_shared_memory_gpu(int height, int width, float* input, unsigned char* output, int length) {
    //Prepare cuda stuff
    float *inputDevice;
    unsigned char * outputDevice;
    hipMalloc((void **) &inputDevice, length);
    hipMalloc((void **) &outputDevice, length / 8);

    hipMemcpy(inputDevice, input, length, hipMemcpyHostToDevice);
    hipMemcpy(outputDevice, output, length / 8, hipMemcpyHostToDevice);

    const dim3 block_size (BLOCK_WIDTH, BLOCK_HEIGHT);
    const dim3 grid_size (GRID_WIDTH, GRID_HEIGHT);

    convertTile<<<grid_size, block_size>>>(height, width, outputDevice, inputDevice);
    printf("Error: %d", hipDeviceSynchronize());

    //Return image to device and free memory
    hipMemcpy(output, outputDevice, length / 8, hipMemcpyDeviceToHost);
    hipFree(inputDevice);
    hipFree(outputDevice);
}
